// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

#include <ff/pasta.hpp>

typedef jacobian_t<vesta_t> point_t;
typedef xyzz_t<vesta_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef pallas_t scalar_t;

#include <msm/pippenger.cuh>
#include <spmvm/spmvm.cuh>

#ifndef __CUDA_ARCH__
extern "C" RustError cuda_pippenger_vesta(point_t *out, const affine_t points[], size_t npoints,
                                          const scalar_t scalars[])
{
    return mult_pippenger<bucket_t>(out, points, npoints, scalars);
}

extern "C" RustError spmvm_vesta(scalar_t out[], const csr_t_host<scalar_t> *csr, const scalar_t scalars[])
{
    return spmvm<scalar_t>(out, csr, scalars);
}

extern "C" RustError spmvm_cpu_vesta(scalar_t out[], const csr_t_host<scalar_t> *csr, const scalar_t scalars[])
{
    return spmvm_cpu<scalar_t>(out, csr, scalars);
}
#endif
