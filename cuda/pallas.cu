// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

#include <ff/pasta.hpp>

typedef jacobian_t<pallas_t> point_t;
typedef xyzz_t<pallas_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef vesta_t scalar_t;

#include <msm/pippenger.cuh>
#include <spmvm/spmvm.cuh>

#ifndef __CUDA_ARCH__

extern "C" void add_test_pallas(void)
{
    add_test();
}

// extern "C" RustError scalar_add_test_pallas(size_t n, const scalar_t *x, const scalar_t *y, scalar_t *out)
// {
//     return scalar_add_test<scalar_t>(n, x, y, out);
// }

extern "C" RustError cuda_pippenger_pallas(point_t *out, const affine_t points[], size_t npoints,
                                           const scalar_t scalars[])
{
    return mult_pippenger<bucket_t>(out, points, npoints, scalars);
}

extern "C" RustError spmvm_pallas(scalar_t out[], const csr_t_host<scalar_t> *csr, const scalar_t scalars[])
{
    return spmvm<scalar_t>(out, csr, scalars);
}

extern "C" RustError spmvm_cpu_pallas(scalar_t out[], const csr_t_host<scalar_t> *csr, const scalar_t scalars[])
{
    return spmvm_cpu<scalar_t>(out, csr, scalars);
}
#endif
